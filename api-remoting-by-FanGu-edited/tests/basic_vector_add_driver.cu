#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// CUDA kernel source code as a string
const char* vectorAddKernel = R"(
extern "C" __global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}
)";

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        const char* errorStr;
        hipDrvGetErrorString(err, &errorStr);
        std::cerr << msg << " failed with error: " << errorStr << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    int N = 1024; // Vector size
    const size_t size = N * sizeof(float);

    // Host vectors
    std::vector<float> h_A(N, 1.0f); // Initialize with 1.0
    std::vector<float> h_B(N, 2.0f); // Initialize with 2.0
    std::vector<float> h_C(N);       // Result vector

    // Initialize CUDA Driver API
    checkCudaError(hipInit(0), "hipInit");

    // Get the first CUDA device
    hipDevice_t device;
    checkCudaError(hipDeviceGet(&device, 0), "hipDeviceGet");

    // Create a CUDA context
    hipCtx_t context;
    checkCudaError(hipCtxCreate(&context, 0, device), "hipCtxCreate");

    // Compile the kernel
    hipModule_t module;
    hipFunction_t kernel;
    checkCudaError(hipModuleLoadDataEx(&module, vectorAddKernel, 0, nullptr, nullptr), "hipModuleLoadDataEx");
    checkCudaError(hipModuleGetFunction(&kernel, module, "vectorAdd"), "hipModuleGetFunction");

    // Allocate device memory
    hipDeviceptr_t d_A, d_B, d_C;
    checkCudaError(hipMalloc(&d_A, size), "hipMalloc for d_A");
    checkCudaError(hipMalloc(&d_B, size), "hipMalloc for d_B");
    checkCudaError(hipMalloc(&d_C, size), "hipMalloc for d_C");

    // Copy host data to device
    checkCudaError(hipMemcpyHtoD(d_A, h_A.data(), size), "hipMemcpyHtoD for d_A");
    checkCudaError(hipMemcpyHtoD(d_B, h_B.data(), size), "hipMemcpyHtoD for d_B");

    // Set up execution configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    void* args[] = { &d_A, &d_B, &d_C, &N };
    checkCudaError(hipModuleLaunchKernel(kernel,
                                  blocksPerGrid, 1, 1,       // Grid dimensions
                                  threadsPerBlock, 1, 1,     // Block dimensions
                                  0, nullptr, args, nullptr), "hipModuleLaunchKernel");

    // Copy result back to host
    checkCudaError(hipMemcpyDtoH(h_C.data(), d_C, size), "hipMemcpyDtoH for d_C");

    // Verify results
    bool success = true;
    for (int i = 0; i < N; ++i) {
        if (h_C[i] != h_A[i] + h_B[i]) {
            success = false;
            std::cerr << "Mismatch at index " << i << ": " << h_C[i] << " != " << h_A[i] + h_B[i] << std::endl;
            break;
        }
    }

    if (success) {
        std::cout << "Vector addition successful (driver version)!" << std::endl;
    }

    // Clean up
    checkCudaError(hipFree(d_A), "hipFree for d_A");
    checkCudaError(hipFree(d_B), "hipFree for d_B");
    checkCudaError(hipFree(d_C), "hipFree for d_C");
    checkCudaError(hipModuleUnload(module), "hipModuleUnload");
    checkCudaError(hipCtxDestroy(context), "hipCtxDestroy");

    return 0;
}