#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    const size_t size = 1024 * sizeof(float);  // Allocate space for 1024 floats
    float* d_ptr = nullptr;
    float* a_ptr = nullptr;
    std::cout << ">>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>" << std::endl;

    // Allocate device memory
    hipError_t err = hipMalloc((void**)&d_ptr, size);
    // if (err != cudaSuccess) {
    //     std::cerr << "cudaMalloc failed: " << cudaGetErrorString(err) << std::endl;
    //     return 1;
    // }

    std::cout << "hipMalloc 1 succeeded. Device pointer: " << d_ptr << std::endl;
    std::cout << ">>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>" << std::endl;

    hipError_t err1 = hipMalloc((void**)&a_ptr, size);
    std::cout << "hipMalloc 2 succeeded. Device pointer: " << a_ptr << std::endl;
    std::cout << ">>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>" << std::endl;

    // Free device memory
    err = hipFree(d_ptr);
    err = hipFree(a_ptr);
    if (err != hipSuccess) {
        std::cerr << "hipFree failed: " << hipGetErrorString(err) << std::endl;
        return 1;
    }

    std::cout << "hipFree succeeded." << std::endl;
    return 0;
}
